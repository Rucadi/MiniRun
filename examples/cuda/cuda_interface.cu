#include "hip/hip_runtime.h"

#include "cuda_interface.hpp"

__global__ void cudaSaxpy(int n, float a, float *x, float *y) 
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)  y[i] = a*x[i] + y[i];
}


void* createStream()
{
    hipStream_t stream;
    hipStreamCreate(&stream);
    return (void*) stream;
}

#include <iostream>
void saxpy(int N, float* d_x, float* d_y, float value, void* stream)
{
    printf("CALLING SAXPY %p %p %p \n", d_x, d_y, stream);
    cudaSaxpy<<<4096,256,0,(hipStream_t) stream>>>(N,value, d_x, d_y);
}

void setActive(int idx)
{
    hipSetDevice(idx);
}


void* cMalloc(size_t size)
{
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}
void cFree(void* ptr)
{
    hipFree(ptr);
}


void copyToDevice(void* dst, void* src, size_t N, void* stream)
{
    hipMemcpyAsync(dst, src, N, hipMemcpyHostToDevice, (hipStream_t) stream);
}
void copyToHost(void* dst, void* src, size_t N, void* stream)
{
    hipMemcpyAsync(dst, src, N, hipMemcpyDeviceToHost, (hipStream_t) stream); 
}

bool streamEmpty(void* stream)
{
    return hipStreamQuery((hipStream_t)  stream) == hipSuccess;
}